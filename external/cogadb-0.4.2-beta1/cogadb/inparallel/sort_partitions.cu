#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <algorithm>
#include <cstdlib>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <moderngpu.cuh>

using namespace std;
using namespace thrust;
using namespace mgpu;

vector<int> generate_unique_data(int num_elements) {

	vector<int> data(num_elements);
        for(unsigned long int j=0; j<num_elements; j++){
	    data[j] = j;
	}
	std::random_shuffle( data.begin(), data.end() );
	return data;
}

vector<int> generate_data(int num_elements) {

	vector<int> data(num_elements);
        for(unsigned long int j=0; j<num_elements; j++){
	    data[j] = std::rand()%num_elements;
	}

	return data;
}


template <int NT, int VT> float blockSortMGPU(vector<int>& host_data, CudaContext &context, int input_size, int numExecutions) {

//template<typename T, typename Comp>
//MGPU_HOST void MergesortKeys(T* data_global, int count, Comp comp,
//    CudaContext& context) {

    device_vector<int> data = host_data;
    int* devptr_input = thrust::raw_pointer_cast(&data[0]);
    
    device_vector<int> output(input_size);;
    int* devptr_output = thrust::raw_pointer_cast(&output[0]); 

    typedef LaunchBoxVT<NT, VT> Tuning;
    int2 launch = Tuning::GetLaunchParams(context);
     
    const int NV = NT * VT;
    
    //cout << "Block length NV: " << NV << endl;
    
    int numBlocks = MGPU_DIV_UP(input_size, NV);
    //int numPasses = FindLog2(numBlocks, true);
 
    //MGPU_MEM(T) destDevice = context.Malloc<T>(count);
    //T* source = data_global;
    //T* dest = destDevice->get();
 
    context.Start();
    
    KernelBlocksort<Tuning, false><<<numBlocks, NT, 0, context.Stream()>>>(devptr_input, (const int*)0, input_size, devptr_output, (int*)0, mgpu::less<int>());
    
    //if(1 & numPasses) std::swap(source, dest);


    float time = context.Split();
    
    cout << input_size / NV << ", " << time << endl;
    
    return time;
}


float sortPartitionsMGPU(vector<int>& host_data, CudaContext &context, int input_size, int num_partitions, int numExecutions) {
    
    
    float time_sum = 0.0f;
    
    for(int i=0; i<numExecutions; i++) {
        device_vector<int> data = host_data;
        int num_elements = input_size / num_partitions;
        context.Start();
        int start = 0;

        while(start < input_size) {
            //end = std::min(input_size, start + num_elements);
            

            int* devptr_input = thrust::raw_pointer_cast(&data[start]);
            mgpu::MergesortKeys(devptr_input, num_elements, mgpu::less<int>(), context);

            
            start += num_elements;
        }
        time_sum += context.Split();
    }
    return time_sum / numExecutions;

}

float sortPartitionsThrust(vector<int>& host_data, CudaContext &context, int input_size, int num_partitions, int numExecutions) {
    
    float time_sum = 0.0f;
    
    for(int i=0; i<numExecutions; i++) {
        device_vector<int> data = host_data;
        int num_elements = input_size / num_partitions;
        context.Start();
        int start = 0;
        int end;
        while(start < input_size) {
            end = std::min(input_size, start + num_elements);
            thrust::sort(data.begin()+start, data.begin()+end);
            start += num_elements;
        }
        time_sum += context.Split();
    }
    return time_sum / numExecutions;

}

template<int NT> void test_partitioned_sort(vector<int>& host_data, CudaContext &context, int size, int numExecutions) {
    cout << "blocksize " << NT << endl;
    
    float time3 = 0;

//    time3 = blockSortMGPU<NT,1>(host_data, context, size, numExecutions);
//
//    time3 = blockSortMGPU<NT,3>(host_data, context, size, numExecutions);
//    
//    time3 = blockSortMGPU<NT,5>(host_data, context, size, numExecutions);
//    
//    time3 = blockSortMGPU<NT,7>(host_data, context, size, numExecutions);
//    
//    time3 = blockSortMGPU<NT,9>(host_data, context, size, numExecutions);
//    
//    time3 = blockSortMGPU<NT,11>(host_data, context, size, numExecutions);
//    
//    time3 = blockSortMGPU<NT,13>(host_data, context, size, numExecutions);
//    
//    time3 = blockSortMGPU<NT,15>(host_data, context, size, numExecutions);
//    
//    time3 = blockSortMGPU<NT,17>(host_data, context, size, numExecutions);
//    
//    time3 = blockSortMGPU<NT,19>(host_data, context, size, numExecutions);
//
//    time3 = blockSortMGPU<NT,21>(host_data, context, size, numExecutions);
//
//    time3 = blockSortMGPU<NT,23>(host_data, context, size, numExecutions);
//    
//    time3 = blockSortMGPU<NT,25>(host_data, context, size, numExecutions);
//        
//    time3 = blockSortMGPU<NT,27>(host_data, context, size, numExecutions);

    //time3 = blockSortMGPU<NT,29>(host_data, context, size, numExecutions);
                
    //time3 = blockSortMGPU<NT,31>(host_data, context, size, numExecutions);

    time3 = blockSortMGPU<NT,32>(host_data, context, size, numExecutions);
    
    time3 = blockSortMGPU<NT,33>(host_data, context, size, numExecutions);
    
    time3 = blockSortMGPU<NT,34>(host_data, context, size, numExecutions);
    
    time3 = blockSortMGPU<NT,35>(host_data, context, size, numExecutions);
    
    time3 = blockSortMGPU<NT,36>(host_data, context, size, numExecutions);
    
    time3 = blockSortMGPU<NT,43>(host_data, context, size, numExecutions);
}

int main(int argc, char** argv) {
	
    cout << "Performance test: Sorting partitions" << endl << endl;
  
    ContextPtr context = CreateCudaDevice(0, 0, true);
    

    
    int size = 2 << 26;
    int numPartitions = 1;
    int numExecutions = 1;
    float time1 = 0;
    float time2 = 0;
    
    vector<int> host_data = generate_data(size);
    
    cout << "Data size is " << (sizeof(int) * size / (2 << 20)) << "MB" << endl;
    
    for(int i = 1; i < 32; i++) {
        
        cout << numPartitions << ", ";

        if(time1 < 6) {
            time1 = sortPartitionsThrust(host_data, *context, size, numPartitions, numExecutions);
            cout << time1;
        }
        cout << ", ";
        
        if(time2 < 6) {
            time2 = sortPartitionsMGPU(host_data, *context, size, numPartitions, numExecutions);
            cout << time2;
        }
        cout << endl;
        
        numPartitions *= 2;
    }
    
    
    test_partitioned_sort<128>(host_data, *context, size, numExecutions);
    
    test_partitioned_sort<256>(host_data, *context, size, numExecutions);
    
    //test_partitioned_sort<386>(host_data, *context, size, numExecutions);
    
    //test_partitioned_sort<512>(host_data, *context, size, numExecutions);
    
    //impossible, too much shared memory
    //test_partitioned_sort<640>(host_data, *context, size, numExecutions);
 
    //test_partitioned_sort<768>(host_data, *context, size, numExecutions);
    
    //test_partitioned_sort<896>(host_data, *context, size, numExecutions);
    
    //test_partitioned_sort<1024>(host_data, *context, size, numExecutions);
    
    return 0;
}

