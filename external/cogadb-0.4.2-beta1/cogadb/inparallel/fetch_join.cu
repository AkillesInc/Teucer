#include "hip/hip_runtime.h"
#include <moderngpu.cuh>
#include <iostream>
#include <vector>
#include <algorithm>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/binary_search.h>
#include <thrust/execution_policy.h>
#include <stdlib.h>     /* srand, rand */
#include <time.h>       /* time */

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %dn", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

using namespace mgpu;
using namespace std;




typedef struct fetch_join_data {
    std::vector<unsigned int> matching_tids;
    std::vector<unsigned int> primary_keys;
    std::vector<unsigned int> foreign_keys;
} fetch_join_data;


void GenerateFetchJoinData(fetch_join_data &data, int join_index_size) {

    //ordered, unique
    std::vector<unsigned int> matching_tids;
    unsigned int num_matching_tids = join_index_size*0.5f;
    //ordered, non-unique
    std::vector<unsigned int> primary_keys;
    float key_repeat_probability = 0.2f;
    //non-ordered unique
    std::vector<unsigned int> foreign_keys;

    int pk = 0;
    for(int i=0; i < join_index_size; i++) {
	foreign_keys.push_back(i);
	primary_keys.push_back(pk);
	float r = static_cast <float> (rand()) / static_cast <float> (RAND_MAX);
	if(r > key_repeat_probability)
	    pk++;
    }
    std::random_shuffle(foreign_keys.begin(), foreign_keys.end());
    
    float prop_use_tid_from_pk_range = (float)num_matching_tids / (float)pk;
    if(prop_use_tid_from_pk_range > 1.0f) {
	std::cout << "Error: More matching tids than different primary keys. Aborting" << endl;
	return;
    }
    
    for(int i=0; i < pk; i++) {
	float r = static_cast <float> (rand()) / static_cast <float> (RAND_MAX);
	if(r < prop_use_tid_from_pk_range)
	    matching_tids.push_back(i);
    }
    num_matching_tids = matching_tids.size();
    cout << "[generated join index (" << join_index_size << " entries) and query tids (" << num_matching_tids << " entries)]" << endl;
  
    data.foreign_keys = foreign_keys;
    data.primary_keys = primary_keys;
    data.matching_tids = matching_tids;
}




__global__ void scatter_fk_bitmap(unsigned int join_index_size, char* flags_in, unsigned int* fk_column_tids, char* flags_out){

    //threads align to join index entries
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    
    while(tid<join_index_size){

        //get foreign key for this thread
	int fk = fk_column_tids[tid];
	
	//scatter to flag array at foreign key index
	if(flags_in[tid])
	    flags_out[fk] = 1;
	
	tid+=blockDim.x * gridDim.x;
    }
}


std::vector<char> DoFetchJoinModernGPU(CudaContext& context, fetch_join_data& data, bool print, bool measure) {
    
    //---- Get sizes ----
  
    unsigned int num_tids = data.matching_tids.size();
    unsigned int join_index_size = data.primary_keys.size();

    //--- Transfer data to device ----
    
    thrust::device_vector<unsigned int> matching_tids(data.matching_tids.begin(), data.matching_tids.end());
    thrust::device_vector<unsigned int> primary_keys(data.primary_keys.begin(), data.primary_keys.end());
    thrust::device_vector<unsigned int> foreign_keys(data.foreign_keys.begin(), data.foreign_keys.end());

    unsigned int *devIn1 = thrust::raw_pointer_cast(&primary_keys[0]);
    unsigned int* devIn2 = thrust::raw_pointer_cast(&matching_tids[0]);
    unsigned int *devIn3 = thrust::raw_pointer_cast(&foreign_keys[0]);
    
    MGPU_MEM(char) out = context.Malloc<char>(primary_keys.size());

    unsigned int number_of_blocks = 512;
    unsigned int number_of_threads = 1024;
    
    thrust::device_ptr<char> flag_array;
    unsigned int number_of_flags = ((join_index_size+7)/8)*8;
    try{
        flag_array = thrust::device_malloc<char>(number_of_flags);
    }catch(std::bad_alloc &e){
        std::cerr << "Ran out of memory during flag array allocation!" << std::endl;
        return std::vector<char>();
    }
    gpuErrchk(hipMemsetAsync(flag_array.get(), 0, number_of_flags));
    hipDeviceSynchronize();

    //--- Perform fetch join
    
    context.Start();
    
    SortedSearch<MgpuBoundsLower, MgpuSearchTypeMatch, MgpuSearchTypeNone>
	(devIn1, primary_keys.size(), devIn2, matching_tids.size(), out->get(), (int*)0, context);
	
    scatter_fk_bitmap<<<number_of_blocks, number_of_threads>>>(join_index_size, out->get(), devIn3, flag_array.get());
    hipDeviceSynchronize();	
	
    double time = context.Split();
    if(measure)
	cout << "ModernGPU SortedSearch + scatter-kernel took " << time << endl << endl;
    
    //---- Transfer results ----

    thrust::host_vector<char> thrust_result(flag_array, flag_array+number_of_flags);
    std::vector<char> result(thrust_result.begin(), thrust_result.end());
    
    //---- Print results
    
    if(print) {
	cout << "--- Results from ModernGPU implementation ---" << endl;
	cout << "join-index (" << join_index_size << ")" << endl;
	for (int i = 0; i < join_index_size; i++)
	    cout << "\t" << data.primary_keys[i];
	cout << endl;
	for (int i = 0; i < join_index_size; i++)
	    cout << "\t" << data.foreign_keys[i];
	cout << endl << "matching-tids (" << num_tids << ")" << endl << "\t";
	for (int i = 0; i < num_tids; i++)
	    cout << ' ' << data.matching_tids[i];
	cout << endl << "flag-array of matching foreign keys (" << number_of_flags << ")" << endl << "\t";
	for (int i = 0; i < number_of_flags; i++)
	    cout << ' ' << (int)flag_array[i];
	cout << endl << endl;
    }
    
    return result;
}


std::vector<char> DoFetchJoinThrust(CudaContext& context, fetch_join_data& data, bool print, bool measure) {

    //---- Get sizes ----
    unsigned int num_tids = data.matching_tids.size();
    unsigned int join_index_size = data.primary_keys.size();

    //--- Transfer data to device ----

    thrust::device_vector<unsigned int> matching_tids(data.matching_tids.begin(), data.matching_tids.end());
    thrust::device_vector<unsigned int> primary_keys(data.primary_keys.begin(), data.primary_keys.end());
    thrust::device_vector<unsigned int> foreign_keys(data.foreign_keys.begin(), data.foreign_keys.end());
    thrust::device_vector<bool> search_output(join_index_size);
    thrust::device_vector<char> bitmap_output(join_index_size);
    thrust::device_vector<char> bits(join_index_size, 1);
    
    //--- Perform fetch join
    
    context.Start();
    
    thrust::binary_search(thrust::device, matching_tids.begin(), matching_tids.end(), primary_keys.begin(), primary_keys.end(), search_output.begin());
    
    thrust::scatter_if(bits.begin(), bits.end(), foreign_keys.begin(), search_output.begin(), bitmap_output.begin());
    
    double time = context.Split();
    if(measure)
	cout << "Thrust implementation took " << time << endl << endl;
    
    //---- Transfer results ----

    thrust::host_vector<char> thrust_result(bitmap_output.begin(), bitmap_output.end());
    std::vector<char> result(thrust_result.begin(), thrust_result.end());
    
    return result;
}




//assumes that matching tids is unique!
__device__ int binary_search(unsigned int* matching_tids, unsigned int number_of_matching_tids, unsigned int search_val){


   int low = 0;
   int high = number_of_matching_tids - 1;
   int mid = low + ((high - low) / 2);

   while (low <= high ){ //&& !(matching_tids[mid - 1] <= search_val && matching_tids[mid] > search_val)) 
       if(mid<0 || mid >= number_of_matching_tids) return number_of_matching_tids;
       //unsigned int mid = low + ((high - low) / 2);

       if (matching_tids[mid] > search_val){
           high = mid - 1;
       }else if (matching_tids[mid] < search_val){
           low = mid + 1;
       }else{
           return (unsigned int) mid; // found
       }
       mid = low + ((high - low) / 2);
   }
 
   return number_of_matching_tids; // not found
}

__global__ void fetch_join_bitmap_kernel(unsigned int* matching_tids, unsigned int number_of_matching_tids, unsigned int* pk_column_tids, unsigned int* fk_column_tids, unsigned int join_index_size, char* flags){

    //threads align to join index entries
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    
    while(tid<join_index_size){

	//try to find primary key for this thread in matching tids
        int index = binary_search(matching_tids, number_of_matching_tids, pk_column_tids[tid]);
	
        //get foreign key for this thread
	int fk = fk_column_tids[tid];
	
	//scatter to flag array at foreign key index
	if(index < number_of_matching_tids)
	    flags[fk] = 1;
	  
	//flags[fk] = index < number_of_matching_tids;
	
	tid+=blockDim.x * gridDim.x;
    }
}

std::vector<char> DoFetchJoinGPU(CudaContext& context, fetch_join_data& data, bool print, bool measure) {

    //---- Get sizes ----
  
    unsigned int num_tids = data.matching_tids.size();
    unsigned int join_index_size = data.primary_keys.size();

    //--- Transfer data to device ----

    thrust::device_vector<unsigned int> device_matching_tids(data.matching_tids.begin(), data.matching_tids.end());
    thrust::device_vector<unsigned int> device_primary_keys(data.primary_keys.begin(), data.primary_keys.end());
    thrust::device_vector<unsigned int> device_foreign_keys(data.foreign_keys.begin(), data.foreign_keys.end());
    
    //---- Perform fetch join ----
    
    unsigned int number_of_blocks = 512;
    unsigned int number_of_threads_per_block = 1024;
    
    thrust::device_ptr<char> flag_array;
    unsigned int number_of_flags = ((join_index_size+7)/8)*8;
    try{
        flag_array = thrust::device_malloc<char>(number_of_flags);
    }catch(std::bad_alloc &e){
        std::cerr << "Ran out of memory during flag array allocation!" << std::endl;
        return std::vector<char>();
    }
    gpuErrchk(hipMemsetAsync(flag_array.get(), 0, number_of_flags));
    hipDeviceSynchronize();
    
    unsigned int* devPtr1 = thrust::raw_pointer_cast(&device_matching_tids[0]);
    unsigned int *devPtr2 = thrust::raw_pointer_cast(&device_primary_keys[0]);
    unsigned int *devPtr3 = thrust::raw_pointer_cast(&device_foreign_keys[0]);
    
    context.Start();
    
    fetch_join_bitmap_kernel<<<number_of_blocks,number_of_threads_per_block>>>(
	devPtr1, num_tids, devPtr2, devPtr3, join_index_size, flag_array.get());
    hipDeviceSynchronize();
    
    double time = context.Split();
    if(measure)
	cout << "GPU Kernel fetch_join_bitmap_kernel took " << time << endl << endl;
    
    //---- Transfer results ----

    thrust::host_vector<char> thrust_result(flag_array, flag_array+number_of_flags);
    std::vector<char> result(thrust_result.begin(), thrust_result.end());
    
    //---- Print results ----
    
    if(print) {
	cout << "--- Results from original GPU kernel ---" << endl;
	cout << "join-index (" << join_index_size << ")" << endl;
	for (int i = 0; i < join_index_size; i++)
	    cout << "\t" << data.primary_keys[i];
	cout << endl;
	for (int i = 0; i < join_index_size; i++)
	    cout << "\t" << data.foreign_keys[i];
	cout << endl << "matching-tids (" << num_tids << ")" << endl << "\t";
	for (int i = 0; i < num_tids; i++)
	    cout << ' ' << data.matching_tids[i];
	cout << endl << "flag-array of matching foreign keys (" << number_of_flags << ")" << endl << "\t";
	for (int i = 0; i < number_of_flags; i++)
	    cout << ' ' << (int)flag_array[i];
	cout << endl << endl;
    }
    
    return result;
}

void TestFetchJoin(CudaContext& context) {
  
    std::vector<char> res1, res2, res3;
  
    int join_index_size = 2 << 20;
    fetch_join_data data;
    GenerateFetchJoinData(data, join_index_size);
    
    res1 = DoFetchJoinGPU(context, data, false, false);
    
    res2 = DoFetchJoinThrust(context, data, false, false);

    res3 = DoFetchJoinModernGPU(context, data, false, false);
    
    bool correct=true;
    for(int i=0; i < join_index_size; i++) {
        char v1,v2,v3;
	v1 = res1.at(i);
	v2 = res2.at(i);
	v3 = res3.at(i);

	if(v1 != v2 | v2 != v3 | v1 != v3) {
	    correct=false;
	    cout << "error at " << i << " aborting test" << endl;
	    return;
	}
    }
    
    if(correct) 
      cout << "Tests were successful. All modules give the same results." << endl << endl;

}



void DoFetchJoin(CudaContext& context) {

    int join_index_size = 16;
    fetch_join_data data;
    GenerateFetchJoinData(data, join_index_size);

    DoFetchJoinGPU(context, data, true, false);

    DoFetchJoinModernGPU(context, data, true, false);
  
}


void DoMeasureFetchJoin(CudaContext& context) {

    int join_index_size = 2 << 26;
    fetch_join_data data;
    GenerateFetchJoinData(data, join_index_size);

    DoFetchJoinGPU(context, data, false, true);
    
    DoFetchJoinThrust(context, data, false, true);

    DoFetchJoinModernGPU(context, data, false, true);
  
}



int main(int argc, char** argv) {
	// Initialize a CUDA device on the default stream.
	ContextPtr context = CreateCudaDevice(argc, argv, true);
	srand (time(NULL));
	
	TestFetchJoin(*context);
	
	DoFetchJoin(*context);

	DoMeasureFetchJoin(*context);
	
	return 0;
}
