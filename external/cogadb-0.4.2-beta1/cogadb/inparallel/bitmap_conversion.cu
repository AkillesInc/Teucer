#include "hip/hip_runtime.h"
#include <moderngpu.cuh>
#include <thrust/scatter.h>
#include <thrust/unique.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/iterator/constant_iterator.h>
#include <iostream>
#include <vector>

using namespace mgpu;
using namespace std;


void PositionListToBitmapCPU(unsigned int* positionlist, unsigned int length, char *flagarray) {
  
    for(int i = 0; i < length; i++) {
	flagarray[positionlist[i]] = 1;
    }

}



vector<char> DoPositionListToBitmapCPU(CudaContext& context, unsigned int positionlist_size, unsigned int table_size, bool print, bool measure) {
	
	std::vector<unsigned int> positionlist;
	std::vector<char> flagarray(table_size);
	
	for(int i = 0; i < positionlist_size; i++)
	    positionlist.push_back(rand() % table_size);
	
	
	context.Start();
	
	PositionListToBitmapCPU(&positionlist[0], positionlist_size, &flagarray[0]);
	
	double time = context.Split();
	if(measure)
	    cout << "CPU PositionlistToBitmap took " << time << endl;
	
	
	if(print) {
	    for (std::vector<unsigned int>::iterator it=positionlist.begin(); it!=positionlist.end(); ++it)
		std::cout << ' ' << *it;
	    cout << endl;
	    for (std::vector<char>::iterator it=flagarray.begin(); it!=flagarray.end(); ++it)
		std::cout << ' ' << (int)*it;
	    cout << endl << endl;
	  
	}
	
	
	return flagarray;
}


__global__ void convertPositionListToBitmap_create_flagarray_kernel(unsigned int* tids, size_t num_tids, size_t num_rows_base_table, char* flag_array){
    
    const int VT=1;
    
    unsigned int tids_start = threadIdx.x*VT + blockIdx.x * blockDim.x;
    unsigned int tids_end = tids_start+VT;
    
    unsigned int thread_tids[VT];
    
    while(tids_end < num_tids) {
        
            for(int i=0; i < VT; i++) {
                thread_tids[i] = tids[tids_start + i];
            }

            for(int i=0; i < VT; i++) {
                flag_array[thread_tids[i]] = 1;
            }
        
        
        tids_start += VT*blockDim.x*gridDim.x;
        tids_end = (tids_start + VT) < num_tids ? tids_start + VT : num_tids;
    }

}


__global__ void convertPositionListToBitmap_create_int_flagarray_kernel(unsigned int* tids, size_t num_tids, size_t num_rows_base_table, int* flag_array){
    
    const int VT=1;
    
    unsigned int tids_start = threadIdx.x*VT + blockIdx.x * blockDim.x;
    unsigned int tids_end = tids_start+VT;
    
    unsigned int thread_tids[VT];
    
    while(tids_end < num_tids) {
        
            for(int i=0; i < VT; i++) {
                thread_tids[i] = tids[tids_start + i];
            }

            for(int i=0; i < VT; i++) {
                flag_array[thread_tids[i]] = 1;
            }
        
        
        tids_start += VT*blockDim.x*gridDim.x;
        tids_end = (tids_start + VT) < num_tids ? tids_start + VT : num_tids;
    }

}


void DoPositionlistToBitmapGPU(CudaContext& context, unsigned int positionlist_size, unsigned int table_size, bool print, bool measure) {
    
    std::vector<unsigned int> positionlist;
    std::vector<char> flagarray(table_size);
    
    for(int i = 0; i < positionlist_size; i++)
	positionlist.push_back(rand() % table_size);
    
    thrust::device_vector<int> device_flagarray(flagarray.begin(), flagarray.end());
    thrust::device_vector<unsigned int> device_positionlist(positionlist.begin(), positionlist.end());
    
    const int number_of_blocks=64;
    const int number_of_threads_per_block=128;
    
    
    unsigned int* devPtr1 = thrust::raw_pointer_cast(&device_positionlist[0]);
    int *devPtr2 = thrust::raw_pointer_cast(&device_flagarray[0]);
    
    context.Start();
    
    convertPositionListToBitmap_create_int_flagarray_kernel<<<number_of_blocks,number_of_threads_per_block>>>(
	    devPtr1, positionlist_size, table_size, devPtr2);
    hipDeviceSynchronize();
    
    double time = context.Split();
    if(measure)
	cout << "GPU Kernel PositionlistToBitmap took " << time << endl;
    
    if(print) {
	for (int i = 0; i < device_positionlist.size(); i++)
	    std::cout << ' ' << device_positionlist[i];
	cout << endl;
	for (int i = 0; i < device_flagarray.size(); i++)
	    std::cout << ' ' << (int)device_flagarray[i];
	cout << endl << endl;
    }



}


void PositionListToBitmapThrust(thrust::device_vector<unsigned int> &positionlist, thrust::device_vector<char> &flag_array) {

    thrust::device_vector<char> bits(positionlist.size(), 1);
  
    thrust::scatter(bits.begin(), bits.end(), positionlist.begin(), flag_array.begin());

}


void DoPositionlistToBitmapThrust(CudaContext& context, unsigned int positionlist_size, unsigned int table_size, bool print, bool measure) {

    std::vector<unsigned int> positionlist;
    std::vector<char> flagarray(table_size);
    
    for(int i = 0; i < positionlist_size; i++)
	positionlist.push_back(rand() % table_size);
    
    thrust::device_vector<char> device_flagarray(flagarray.begin(), flagarray.end());
    thrust::device_vector<unsigned int> device_positionlist(positionlist.begin(), positionlist.end());
    
    context.Start();
    
    PositionListToBitmapThrust(device_positionlist, device_flagarray);
	
    double time = context.Split();
    if(measure)
	cout << "Thrust PositionlistToBitmap took " << time << endl;
    
    if(print) {
	for (int i = 0; i < device_positionlist.size(); i++)
	    std::cout << ' ' << device_positionlist[i];
	cout << endl;
	for (int i = 0; i < device_flagarray.size(); i++)
	    std::cout << ' ' << (int)device_flagarray[i];
	cout << endl << endl;
    }
}



void PositionListToBitmapModernGPU(CudaContext& context, thrust::device_vector<unsigned int> &positionlist, thrust::device_vector<char> &flag_array) {

      unsigned int *devIn1 = thrust::raw_pointer_cast(&positionlist[0]);

      MergesortKeys(devIn1, positionlist.size(), mgpu::less<int>(), context);



}


void DoPositionlistToBitmapModernGPU(CudaContext& context, unsigned int positionlist_size, unsigned int table_size, bool print, bool measure) {

    std::vector<unsigned int> positionlist;
    std::vector<char> flagarray(table_size);
    
    for(int i = 0; i < positionlist_size; i++)
	positionlist.push_back(rand() % table_size);
    
    thrust::device_vector<char> device_flagarray(flagarray.begin(), flagarray.end());
    thrust::device_vector<unsigned int> device_positionlist(positionlist.begin(), positionlist.end());
    
    context.Start();
    
    PositionListToBitmapModernGPU(context, device_positionlist, device_flagarray);
	
    double time = context.Split();
    if(measure)
	cout << "ModernGPU PositionlistToBitmap took " << time << endl;
    
    if(print) {
	for (int i = 0; i < device_positionlist.size(); i++)
	    std::cout << ' ' << device_positionlist[i];
	cout << endl;
	for (int i = 0; i < device_flagarray.size(); i++)
	    std::cout << ' ' << (int)device_flagarray[i];
	cout << endl << endl;
    }
}



void DoPositionlistToBitmap(CudaContext& context) {

    int positionlist_size = 2 << 4;
    int table_size = 2 << 5;

    DoPositionListToBitmapCPU(context, positionlist_size, table_size, true, false);
	
    DoPositionlistToBitmapThrust(context, positionlist_size, table_size, true, false);

    DoPositionlistToBitmapGPU(context, positionlist_size, table_size, true, false);
}


void DoMeasurePositionlistToBitmap(CudaContext& context) {

    int positionlist_size = 2 << 25;
    int table_size = 2 << 26;

    DoPositionListToBitmapCPU(context, positionlist_size, table_size, false, true);
	
    DoPositionlistToBitmapThrust(context, positionlist_size, table_size, false, true);
  
    DoPositionlistToBitmapGPU(context, positionlist_size, table_size, false, true);
    
    DoPositionlistToBitmapModernGPU(context, positionlist_size, table_size, false, true);
}





template<int NT, typename InputIt1, typename InputIt2, typename OutputIt,
    typename Comp>
__global__ void ParallelMergeA(InputIt1 a_global, int aCount, InputIt2 b_global,
    int bCount, OutputIt dest_global, Comp comp) {
 
    typedef typename std::iterator_traits<InputIt1>::value_type T;
 
    int gid = threadIdx.x + NT * blockIdx.x;
    if(gid < aCount) {
        T aKey = a_global[gid];
        int lb = BinarySearch<MgpuBoundsLower>(b_global, bCount, aKey, comp);
        dest_global[gid + lb] = aKey;
    }
}
 
template<int NT, typename InputIt1, typename InputIt2, typename OutputIt,
    typename Comp>
__global__ void ParallelMergeB(InputIt1 a_global, int aCount, InputIt2 b_global,
    int bCount, OutputIt dest_global, Comp comp) {
 
    typedef typename std::iterator_traits<InputIt2>::value_type T;
 
    int gid = threadIdx.x + NT * blockIdx.x;
    if(gid < bCount) {
        T bKey = b_global[gid];
        int ub = BinarySearch<MgpuBoundsUpper>(a_global, aCount, bKey, comp);
        dest_global[gid + ub] = bKey;
    }
}


void DoMerge(CudaContext& context) {
	printf("\n\nMERGE DEMONSTRATION\n\n");

	int ACount = 30;
	int BCount = 30;

	MGPU_MEM(int) aKeysDevice = context.SortRandom<int>(ACount, 100, 130);
	MGPU_MEM(int) bKeysDevice = context.SortRandom<int>(BCount, 100, 130);
	MGPU_MEM(int) cKeysDevice = context.Malloc<int>(ACount+BCount);

	printf("A keys:\n");
	PrintArray(*aKeysDevice, "%4d", 10);

	printf("\nB keys:\n");
	PrintArray(*bKeysDevice, "%4d", 10);
	
	const int NT = 512;
	int ablocks = MGPU_DIV_UP(ACount, NT);
	int bblocks = MGPU_DIV_UP(BCount, NT);
	ParallelMergeA<NT><<<ablocks, NT>>>(aKeysDevice->get(), ACount, bKeysDevice->get(), BCount, cKeysDevice->get(), mgpu::less<int>());
	ParallelMergeB<NT><<<bblocks, NT>>>(aKeysDevice->get(), ACount, bKeysDevice->get(), BCount, cKeysDevice->get(), mgpu::less<int>());

	printf("\nC keys:\n");
	PrintArray(*cKeysDevice, "%4d", 10);
}


void DoJoin(CudaContext& context) {
	printf("\n\nRELATIONAL JOINS DEMONSTRATION\n\n");

	int ACount = 30;
	int BCount = 30;

	MGPU_MEM(int) aKeysDevice = context.SortRandom<int>(ACount, 100, 130);
	MGPU_MEM(int) bKeysDevice = context.SortRandom<int>(BCount, 100, 130);
	std::vector<int> aKeysHost, bKeysHost;
	aKeysDevice->ToHost(aKeysHost);
	bKeysDevice->ToHost(bKeysHost);

	printf("A keys:\n");
	PrintArray(*aKeysDevice, "%4d", 10);

	printf("\nB keys:\n");
	PrintArray(*bKeysDevice, "%4d", 10);

	MGPU_MEM(int) aIndices, bIndices;
	int innerCount = RelationalJoin<MgpuJoinKindInner>(aKeysDevice->get(),
		ACount, bKeysDevice->get(), BCount, &aIndices, &bIndices, context);

	std::vector<int> aHost, bHost;
	aIndices->ToHost(aHost);
	bIndices->ToHost(bHost);

	printf("\nInner-join (%d items):\n", innerCount);
	printf("output   (aIndex, bIndex) : (aKey, bKey)\n");
	printf("----------------------------------------\n");
	for(int i = 0; i < innerCount; ++i)
		printf("%3d      (%6d, %6d) : (%4d, %4d)\n", i, aHost[i], bHost[i],
			aKeysHost[aHost[i]], bKeysHost[bHost[i]]);

	int outerCount = RelationalJoin<MgpuJoinKindOuter>(aKeysDevice->get(),
		ACount, bKeysDevice->get(), BCount, &aIndices, &bIndices, context);

	aIndices->ToHost(aHost);
	bIndices->ToHost(bHost);
	printf("\nOuter-join (%d items):\n", outerCount);
	printf("output   (aIndex, bIndex) : (aKey, bKey)\n");
	printf("----------------------------------------\n");
	for(int i = 0; i < outerCount; ++i) {
		std::string aKey, bKey;
		if(-1 != aHost[i]) aKey = stringprintf("%4d", aKeysHost[aHost[i]]);
		if(-1 != bHost[i]) bKey = stringprintf("%4d", bKeysHost[bHost[i]]);
		printf("%3d      (%6d, %6d) : (%4s, %4s)\n", i, aHost[i], bHost[i],
			(-1 != aHost[i]) ? aKey.c_str() : "---", 
			(-1 != bHost[i]) ? bKey.c_str() : "---");
	}
}





int main(int argc, char** argv) {
	// Initialize a CUDA device on the default stream.
	ContextPtr context = CreateCudaDevice(argc, argv, true);
	
	DoPositionlistToBitmap(*context);

	DoMeasurePositionlistToBitmap(*context);
	
	return 0;
}
